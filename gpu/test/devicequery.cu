#include "hip/hip_runtime.h"
#include ""
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
int sample2()
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int dev;
	for (dev = 0; dev < deviceCount; dev++)
	{
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		if (dev == 0)
		{
			if (/*deviceProp.major==9999 && */deviceProp.minor = 9999 && deviceProp.major == 9999)
				printf("\n");

		}
		printf("\nDevice%d:\"%s\"\n", dev, deviceProp.name);
		printf("Total amount of global memory                   %u bytes\n", deviceProp.totalGlobalMem);
		printf("Number of mltiprocessors                        %d\n", deviceProp.multiProcessorCount);
		printf("Total amount of constant memory:                %u bytes\n", deviceProp.totalConstMem);
		printf("Total amount of shared memory per block         %u bytes\n", deviceProp.sharedMemPerBlock);
		printf("Total number of registers available per block:  %d\n", deviceProp.regsPerBlock);
		printf("Warp size                                       %d\n", deviceProp.warpSize);
		printf("Maximum number of threada per block:            %d\n", deviceProp.maxThreadsPerBlock);
		printf("Maximum sizes of each dimension of a block:     %d x %d x %d\n", deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf("Maximum size of each dimension of a grid:       %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
		printf("Maximum memory pitch :                          %u bytes\n", deviceProp.memPitch);
		printf("Texture alignmemt                               %u bytes\n", deviceProp.texturePitchAlignment);
		printf("Clock rate                                      %.2f GHz\n", deviceProp.clockRate*1e-6f);
	}
	printf("\nTest PASSED\n");
	getchar();
	return 0;
}